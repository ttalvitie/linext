#include "hip/hip_runtime.h"
#include "relaxtpa_gpu.hpp"

#include "pcg32.hpp"

namespace {

const int MaxCoord = (int)(((unsigned int)1 << 31) - 1);
const ll InitialIterationCount = 16;
const ll initialFinalWalkCount = (ll)1 << 62;

__device__ ll signedAtomicAdd(ll& var, ll val) {
    return (ll)atomicAdd((ull*)&var, (ull)val);
}

struct TPAState {
    ll prelimWalkCount;
    const ll* finalWalkCountTable;
    ll finalWalkCountTableSize;
    ll finalWalkCountTarget;
    ll finalWalkCount;
    ll prelimResults;
    ll nextWalkIdx;
    ll finalHitCount;
    ll totalIterCount;

    __device__ bool popWalkJob(bool& prelim) {
        ll walkIdx = signedAtomicAdd(nextWalkIdx, -1);
        if(walkIdx >= 0) {
            prelim = walkIdx >= initialFinalWalkCount;
            return true;
        } else {
            return false;
        }
    }

    __device__ void pushWalkResult(bool prelim, ll hitCount) {
        if(prelim) {
            // prelimResults stores number of results in top 16 bits
            ll increment = ((ll)1 << 48) | hitCount;
            ll results = signedAtomicAdd(prelimResults, increment);
            results += increment;

            ll prelimResultCount = results >> 48;
            if(prelimResultCount == prelimWalkCount) {
                ll prelimHitCount = results & (((ll)1 << 48) - (ll)1);
                ll prelimEstimateCeil = (prelimHitCount + prelimWalkCount - 1) / prelimWalkCount;
                prelimEstimateCeil = min(prelimEstimateCeil, finalWalkCountTableSize - 1);

                finalWalkCountTarget = finalWalkCountTable[prelimEstimateCeil];
                finalWalkCount = finalWalkCountTarget;

                ll decrement = initialFinalWalkCount - finalWalkCount;
                ll idx = signedAtomicAdd(nextWalkIdx, -decrement);
                idx -= decrement;

                if(idx < -1) {
                    finalWalkCount += -1 - idx;
                }
            }
        } else {
            signedAtomicAdd(finalHitCount, hitCount);
        }
    }

    double prelimEstimate() const {
        ll prelimHitCount = prelimResults & (((ll)1 << 48) - (ll)1);
        return (double)prelimHitCount / (double)prelimWalkCount;
    }

    double finalEstimate() const {
        return (double)finalHitCount / (double)finalWalkCount;
    }
};

#include "relaxtpa_gpu_kernel.cuh"
#define VERT_POS_USE_SHARED_MEM
#include "relaxtpa_gpu_kernel.cuh"
#undef VERT_POS_USE_SHARED_MEM

template <typename T>
struct GPUDeleter {
    void operator()(T* ptr) {
        CUDACHECK(hipFree(ptr));
    }
};

template <typename T>
using GPUArray = unique_ptr<T[], GPUDeleter<T>>;

template <typename T>
GPUArray<T> createUninitializedGPUArray(int elemCount) {
    T* ptrDev;
    size_t size = sizeof(T) * elemCount;
    CUDACHECK(hipMalloc(&ptrDev, size));
    return GPUArray<T>(ptrDev);
}

template <typename T>
GPUArray<T> vecToGPU(const vector<T>& vec) {
    T* ptrDev;
    size_t size = sizeof(T) * vec.size();
    CUDACHECK(hipMalloc(&ptrDev, size));
    CUDACHECK(hipMemcpy(ptrDev, vec.data(), size, hipMemcpyHostToDevice));
    return GPUArray<T>(ptrDev);
}

template <typename T>
using GPUStruct = unique_ptr<T, GPUDeleter<T>>;

template <typename T>
GPUStruct<T> structToGPU(const T& src) {
    T* ptrDev;
    CUDACHECK(hipMalloc(&ptrDev, sizeof(T)));
    CUDACHECK(hipMemcpy(ptrDev, &src, sizeof(T), hipMemcpyHostToDevice));
    return GPUStruct<T>(ptrDev);
}

template <typename T>
void structFromGPU(const GPUStruct<T>& gpuStruct, T& dest) {
    CUDACHECK(hipMemcpy(&dest, gpuStruct.get(), sizeof(T), hipMemcpyDeviceToHost));
}

hipDeviceProp_t getGPUProp() {
    int device;
    CUDACHECK(hipGetDevice(&device));
    hipDeviceProp_t gpuProp;
    CUDACHECK(hipGetDeviceProperties(&gpuProp, device));
    return gpuProp;
}

}

void runRelaxTpaGPU(
    int n,
    double relaxationLogCount,
    const vector<ll>& finalWalkCountTable,
    const vector<vector<uint32_t>>& warpOps,
    ll prelimWalkCount
) {
    hipDeviceProp_t gpuProp = getGPUProp();
    msg("GPU_SM_COUNT ", gpuProp.multiProcessorCount);
    msg("GPU_SHARED_MEM_SIZE ", gpuProp.sharedMemPerMultiprocessor);
    int blockCount = gpuProp.multiProcessorCount;

    const int vertPosSizePerBlock = 32 * max(n, 1);

    msg("GPU_VERT_POS_BYTES_PER_BLOCK ", vertPosSizePerBlock * sizeof(int));
    bool useSharedMem;
    if(vertPosSizePerBlock * sizeof(int) <= gpuProp.sharedMemPerMultiprocessor) {
        msg("GPU_VERT_POS_USE_SHARED_MEM");
        useSharedMem = true;
    } else {
        msg("GPU_VERT_POS_USE_GLOBAL_MEM");
        useSharedMem = false;
    }

    GPUArray<ll> finalWalkCountTableDev = vecToGPU(finalWalkCountTable);

    vector<uint32_t> warpOpData;
    vector<int> warpOpOffsets(WarpCount);
    for(int warpIdx = 0; warpIdx < WarpCount; ++warpIdx) {
        warpOpOffsets[warpIdx] = warpOpData.size();
        for(uint32_t op : warpOps[warpIdx]) {
            warpOpData.push_back(op);
        }
        while(warpOpData.size() & 31) {
            warpOpData.push_back(0);
        }
    }
    for(int i = 0; i < 32; ++i) {
        warpOpData.push_back(0);
    }

    GPUArray<uint32_t> warpOpDataDev = vecToGPU(warpOpData);

    vector<const uint32_t*> warpOpStarts;
    for(int offset : warpOpOffsets) {
        warpOpStarts.push_back(warpOpDataDev.get() + offset);
    }

    GPUArray<const uint32_t*> warpOpStartsDev = vecToGPU(warpOpStarts);

    vector<int> warpRandsPerIteration(WarpCount);
    for(int warpIdx = 0; warpIdx < WarpCount; ++warpIdx) {
        int count = 0;
        for(uint32_t op : warpOps[warpIdx]) {
            if((op & 0xF000) == 0x8000) {
                ++count;
            }
            if(((op >> 16) & 0xF000) == 0x8000) {
                ++count;
            }
        }
        warpRandsPerIteration[warpIdx] = count;
    }

    GPUArray<int> warpRandsPerIterationDev = vecToGPU(warpRandsPerIteration);

    TPAState tpaState;
    tpaState.prelimWalkCount = prelimWalkCount;
    tpaState.finalWalkCountTable = finalWalkCountTableDev.get();
    tpaState.finalWalkCountTableSize = finalWalkCountTable.size();
    tpaState.finalWalkCountTarget = -1;
    tpaState.finalWalkCount = -1;
    tpaState.prelimResults = 0;
    tpaState.nextWalkIdx = initialFinalWalkCount + prelimWalkCount - 1;
    tpaState.finalHitCount = 0;
    tpaState.totalIterCount = 0;
    
    GPUStruct<TPAState> tpaStateDev = structToGPU(tpaState);

    GPUArray<int> vertPosBufDev;
    if(useSharedMem) {
        CUDACHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel_shared),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            vertPosSizePerBlock * sizeof(int)
        ));
    } else {
        CUDACHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel_global),
            hipFuncAttributePreferredSharedMemoryCarveout,
            cudaSharedmemCarveoutMaxL1
        ));

        vertPosBufDev = createUninitializedGPUArray<int>(vertPosSizePerBlock * blockCount);
    }

    uint64_t seed = UnifInt<uint64_t>()(rng);

    msg("PARALLEL_TPA_START");

    msg("GPU_SAMPLE_KERNEL_BLOCK_COUNT ", blockCount);
    msg("GPU_SAMPLE_KERNEL_BLOCK_SIZE ", 32 * WarpCount);
    msg("GPU_SAMPLE_KERNEL_START");

    if(useSharedMem) {
        kernel_shared<<<blockCount, 32 * WarpCount, vertPosSizePerBlock * sizeof(int)>>>(
            n,
            seed,
            warpOpStartsDev.get(),
            warpRandsPerIterationDev.get(),
            tpaStateDev.get()
        );
    } else {
        kernel_global<<<blockCount, 32 * WarpCount>>>(
            n,
            seed,
            warpOpStartsDev.get(),
            warpRandsPerIterationDev.get(),
            tpaStateDev.get(),
            vertPosBufDev.get()
        );
    }

    CUDACHECK(hipGetLastError());
    CUDACHECK(hipDeviceSynchronize());
    msg("GPU_SAMPLE_KERNEL_END");

    structFromGPU(tpaStateDev, tpaState);

    msg("PARALLEL_TPA_END");
    msg("PARALLEL_TPA_PRELIM_ESTIMATE ", tpaState.prelimEstimate());
    msg("PARALLEL_TPA_FINAL_WALK_COUNT_TARGET ", tpaState.finalWalkCountTarget);
    msg("PARALLEL_TPA_FINAL_WALK_COUNT ", tpaState.finalWalkCount);
    msg("PARALLEL_TPA_FINAL_ESTIMATE ", tpaState.finalEstimate());
    msg("TOTAL_ITER_COUNT ", tpaState.totalIterCount);
    msg("LINEXT_LOG_COUNT ", relaxationLogCount - tpaState.finalEstimate());
}
